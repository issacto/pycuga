
#include <hip/hip_runtime.h>
__global__ void evaluation(unsigned long long int *parents, int ulonglongRequired, unsigned int *chromosomesResults, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    chromosomesResults[(blockIdx.x * blockDim.x + threadIdx.x)]=1;
}