
#include <hip/hip_runtime.h>
__global__ void evaluation(unsigned long long int *parents, int ulonglongRequired, unsigned int *chromosomesResults, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    chromosomesResults[(blockIdx.x * blockDim.x + threadIdx.x)]=1;
    if((id%32)==0){
        chromosomesResults[(blockIdx.x * blockDim.x + threadIdx.x)]=2;
    }
}