
#include <hip/hip_runtime.h>
__global__ void selection_elitism(unsigned long long int *parents, int ulonglongRequired,  int *parentVals, unsigned long long int *blockBestParent, int islandSize, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * islandSize;
        int tmpLargestVal = 0;
        int tmpLargestPar = 0;
        // iterate over the threads in an island
        for (int i = 0; i < islandSize; i++)
        {
            if (parentVals[bId + i] > tmpLargestVal)
            {
                tmpLargestPar = (bId + i)*ulonglongRequired;
                tmpLargestVal = parentVals[bId + i];
            }
        }
        // select the chromosome with the highest fitness value at the corresponding blockBestParent array
        for(int i = 0 ;i<ulonglongRequired;i++){
            blockBestParent[id*ulonglongRequired+i] = parents[tmpLargestPar+i];
        }
    }
}

__global__ void selection_roulettewheel(unsigned long long int *parents, int ulonglongRequired,  int *parentVals, unsigned long long int *blockBestParent, float *wheelProbs, int islandSize, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * islandSize;
        unsigned int tmpLowestVal = 100000000;
        unsigned int totalVal = 0;
        // find the lowest and total fitness value
        for (int i = 0; i < islandSize; i++)
        {
            if (parentVals[bId + i] < tmpLowestVal)
            {
                tmpLowestVal = parentVals[bId + i];
            }
            totalVal += parentVals[bId + i];
        }
        unsigned int base = totalVal - islandSize * tmpLowestVal;
        // store the cumulative proabability
        float tmpProb = 0;
        for (int i = 0; i < islandSize; i++)
        {
            tmpProb += (parentVals[bId + i] - tmpLowestVal) / base;
            if (tmpProb > wheelProbs[id])
            {
                // select the chromosome when the probability is higher than the randomly generated probability
                blockBestParent[id] = parents[bId + i];
            }
        }
    }
}
