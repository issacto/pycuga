#include "hip/hip_runtime.h"
__global__ void selection_elitism(unsigned long long int *parents, int ulonglongRequired, unsigned int *parentVals, unsigned long long int *blockBestParent, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * threadsInBlockIsland;
        unsigned int tmpLargestIndex = 0;
        unsigned long long int tmpLargestPar = 0;
        // iterate over the threads in an island
        for (int i = 0; i < threadsInBlockIsland; i++)
        {
            if (parentVals[bId + i] > tmpLargestVal)
            {
                tmpLargestPar = parents[bId + i];
                tmpLargestIndex = parentVals[bId + i];
            }
        }
        // select the chromosome with the highest fitness value at the corresponding blockBestParent array
        for(int i = 0 ;i<ulonglongRequired;i++){
            blockBestParent[id*ulonglongRequired+i] = blockBestParent[ulonglongRequired*ulonglongRequired+i];
        }
    }
}

