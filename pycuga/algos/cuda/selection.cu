
#include <hip/hip_runtime.h>
__global__ void selection(unsigned long long int *parents, int ulonglongRequired,  int *parentVals, unsigned long long int *blockBestParent, int islandSize, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * islandSize;
        int tmpLargestVal = 0;
        int tmpLargestPar = 0;
        // iterate over the threads in an island
        for (int i = 0; i < islandSize; i++)
        {
            if (parentVals[bId + i] > tmpLargestVal)
            {
                tmpLargestPar = (bId + i)*ulonglongRequired;
                tmpLargestVal = parentVals[bId + i];
            }
        }
        // select the chromosome with the highest fitness value at the corresponding blockBestParent array
        for(int i = 0 ;i<ulonglongRequired;i++){
            blockBestParent[id*ulonglongRequired+i] = parents[tmpLargestPar+i];
        }
    }
}

