#include "hip/hip_runtime.h"

__global__ void internalReOrder(unsigned long long int *parents, int ulonglongRequired, unsigned int *parentVals, int islandSize, int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        int bId = id * islandSize;
        int lowestIndex, highestIndex, highestVal = 0;
        int lowestVal = 2147483647;
        ;
        for (int i = 0; i < islandSize; i++)
        {
            // store the chromsomes with the lowest and highest fitness values
            if (i == 0)
            {
                lowestVal = parentVals[bId + i];
                highestVal = parentVals[bId + i];
                lowestIndex = bId + i;
                highestIndex = bId + i;
            }
            else
            {
                if (parentVals[bId + i] < lowestVal)
                {
                    lowestVal = parentVals[bId + i];
                    lowestIndex = bId + i;
                }
                else if (parentVals[bId + i] > highestVal)
                {
                    highestVal = parentVals[bId + i];
                    highestIndex = bId + i;
                }
            }
        }
        unsigned long long int tmpLowest[ULONGLONGREQUIREDVALUE];
        unsigned long long int tmpHighest[ULONGLONGREQUIREDVALUE];
        //TODO
        for(int i =0; i < ulonglongRequired ; i++){
            tmpLowest[i]=parents[lowestIndex*ulonglongRequired+i];
        }
        for(int i =0; i < ulonglongRequired ; i++){
            tmpHighest[i]=parents[highestIndex*ulonglongRequired+i];
        }
        // swap the position of the first position with that of the chromosome with lowest fitness values
        for(int i =0; i < ulonglongRequired ; i++){
            parents[lowestIndex*ulonglongRequired+i] = parents[bId*ulonglongRequired+i];
        }
        for(int i =0; i < ulonglongRequired ; i++){
            parents[bId*ulonglongRequired+i]=tmpLowest[i];
        }
        // swap the position of the last position with that of the chromosome with highest fitness values
        for(int i =0; i < ulonglongRequired ; i++){
            parents[highestIndex*ulonglongRequired+i] = parents[(bId + islandSize-1)*ulonglongRequired+i];
        }
        for(int i =0; i < ulonglongRequired ; i++){
            parents[(bId + islandSize-1)*ulonglongRequired+i]=tmpHighest[i];
        }
    }
}

__global__ void migration(unsigned long long int *parents, int ulonglongRequired, int islandSize , int max)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (max > id)
    {
        // Migration - the last chromosome replaces the first chromosome of the next block
        int index = ((id + 1) * islandSize - 1)*ulonglongRequired;
        if (index >= max)  index = index - max;
        int replaceIndex = ((id + 1) * islandSize)*ulonglongRequired;
        if (replaceIndex >= max)  replaceIndex = replaceIndex - max;
        for(int i =0; i < ulonglongRequired ; i++){
            parents[replaceIndex+i]=parents[index+i];
        }
    }
}
