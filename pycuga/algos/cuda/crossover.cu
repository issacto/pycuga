
#include <hip/hip_runtime.h>

__global__ void crossover_fixed(unsigned long long int *parents, int ulonglongRequired, unsigned long long int *blockBestParents, int *splitIndex, int *length, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    int startingPosition = splitIndex[id] - length[id];
    if (startingPosition < 0)   startingPosition = 0;
    int startingBlock = 0;
    if (startingPosition != 0) startingBlock = startingPosition/64;
    int startingIndex = startingPosition-64*startingBlock;
    
   
    if (max > id)
    {
        int bId = blockIdx.x * ulonglongRequired;
        for(int i = startingBlock; i< ulonglongRequired;i++ ){
            for (int ii = startingIndex; ii < 64; ii++)
            {
                if ((blockBestParents[bId+i] >> ii) & 1)
                {
                    // if selected chromsome ith index is 1
                    if (!((parents[id+i] >> ii) & 1))
                    {
                        // if chromsome ith index is 0
                        parents[id+i] |= (1ULL << ii);
                    }
                }
                else
                {
                    // if selected chromsome ith index is 0
                    if ((parents[id+i] >> ii) & 1)
                    {
                        // if chromsome ith index is 1
                        parents[id+i] &= ~(1ULL << ii);
                    }
                }
            }
        }
        startingIndex =0;
    }
}

__global__ void crossover_uniform(unsigned long long int *parents, int ulonglongRequired, unsigned long long int *blockBestParents, int *splitIndex, int *length, int max)
{
    int id =(blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    if (max > id)
    {
        int bId = blockIdx.x;
        for(int i = id; i< ulonglongRequired;i++ ){
            for (int ii = 0; ii < 64; ii += 2)
            {
                if ((blockBestParents[bId+i] >> ii) & 1)
                {
                    // if selected chromsome ith index is 1
                    if (!((parents[id+i] >> ii) & 1))
                    {
                        // if chromsome ith index is 0
                        parents[id+i] |= (1ULL << ii);
                    }
                }
                else
                {
                    // if selected chromsome ith index is 0
                    if ((parents[id+i] >> ii) & 1)
                    {
                        // if chromsome ith index is 1
                        parents[id+i] &= ~(1ULL << ii);
                    }
                }
            }
        }
    }
}
