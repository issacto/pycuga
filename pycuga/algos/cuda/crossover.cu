
#include <hip/hip_runtime.h>

__global__ void crossover_single(unsigned long long int *parents, int ulonglongRequired, unsigned long long int *blockBestParents, int *splitIndex, int islandSize, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    int startingPosition = splitIndex[(blockIdx.x * blockDim.x + threadIdx.x)];
    if (startingPosition < 0)   startingPosition = 0;
    int startingBlock = 0;
    if (startingPosition != 0) startingBlock = startingPosition/64;
    int startingIndex = startingPosition-64*startingBlock;
    
   
    if (max > id)
    {
        int bId = (blockIdx.x * blockDim.x + threadIdx.x)/islandSize;
        for(int i = startingBlock; i< ulonglongRequired;i++ ){
            for (int ii = startingIndex; ii < 64; ii++)
            {
                if ((blockBestParents[bId+i] >> ii) & 1)
                {
                    // if selected chromsome ith index is 1
                    if (!((parents[id+i] >> ii) & 1))
                    {
                        // if chromsome ith index is 0
                        parents[id+i] |= (1ULL << ii);
                    }
                }
                else
                {
                    // if selected chromsome ith index is 0
                    if ((parents[id+i] >> ii) & 1)
                    {
                        // if chromsome ith index is 1
                        parents[id+i] &= ~(1ULL << ii);
                    }
                }
            }
        }
        startingIndex =0;
    }
}

__global__ void crossover_double(unsigned long long int *parents, int ulonglongRequired, unsigned long long int *blockBestParents, int *splitIndex, int *length, int islandSize, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    int startingPosition = splitIndex[(blockIdx.x * blockDim.x + threadIdx.x)];
    if (startingPosition < 0)   startingPosition = 0;
    int startingBlock = 0;
    if (startingPosition != 0) startingBlock = startingPosition/64;
    int startingIndex = startingPosition%64;
    int endingBlock =(startingPosition+length[(blockIdx.x * blockDim.x + threadIdx.x)])/64+1;
    int endingIndex = (startingPosition+length[(blockIdx.x * blockDim.x + threadIdx.x)])%64+1;
    if((startingPosition+length[(blockIdx.x * blockDim.x + threadIdx.x)])>ulonglongRequired*64){
        endingBlock=ulonglongRequired;
        endingIndex = 64;
    }
   
    if (max > id)
    {
        int bId = (blockIdx.x * blockDim.x + threadIdx.x)/islandSize;
        for(int i = startingBlock; i< endingBlock;i++ ){
            for (int ii = startingIndex; ii < endingIndex; ii++)
            {
                if ((blockBestParents[bId+i] >> ii) & 1)
                {
                    // if selected chromsome ith index is 1
                    if (!((parents[id+i] >> ii) & 1))
                    {
                        // if chromsome ith index is 0
                        parents[id+i] |= (1ULL << ii);
                    }
                }
                else
                {
                    // if selected chromsome ith index is 0
                    if ((parents[id+i] >> ii) & 1)
                    {
                        // if chromsome ith index is 1
                        parents[id+i] &= ~(1ULL << ii);
                    }
                }
            }
        }
        startingIndex =0;
    }
}

__global__ void crossover_uniform(unsigned long long int *parents, int ulonglongRequired, unsigned long long int *blockBestParents, int *splitIndex, int *length, int max)
{
    int id =(blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    if (max > id)
    {
        int bId = blockIdx.x;
        for(int i = id; i< ulonglongRequired;i++ ){
            for (int ii = 0; ii < 64; ii += 2)
            {
                if ((blockBestParents[bId+i] >> ii) & 1)
                {
                    // if selected chromsome ith index is 1
                    if (!((parents[id+i] >> ii) & 1))
                    {
                        // if chromsome ith index is 0
                        parents[id+i] |= (1ULL << ii);
                    }
                }
                else
                {
                    // if selected chromsome ith index is 0
                    if ((parents[id+i] >> ii) & 1)
                    {
                        // if chromsome ith index is 1
                        parents[id+i] &= ~(1ULL << ii);
                    }
                }
            }
        }
    }
}
