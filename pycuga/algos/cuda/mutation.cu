
#include <hip/hip_runtime.h>
// __global__ void mutation(unsigned long long int *parents, int ulonglongRequired, int *mutateIndex, int max)
// {
//     int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
//     if (max > id)
//     {

//         int mutateIndexId = mutateIndex[blockIdx.x * blockDim.x + threadIdx.x] / 64 +id;
//         int mutateDigit =  mutateIndex[blockIdx.x * blockDim.x + threadIdx.x] % 64;
//         // if(mutateIndexId<max  && mutateDigit<64 && mutateDigit>=0){
//             int tmpVar = parents[mutateIndexId];
//             if (!((tmpVar >> mutateDigit) & 1))
//             {
//                 tmpVar |= (1ULL << mutateDigit);
//             }
//             else
//             {
//                 // if chromsome idth index is 1
//                 tmpVar &= ~(1ULL << mutateDigit);
//             }
//             parents[mutateIndexId]=  tmpVar;
//         // }

//     }
// }


__global__ void mutation(unsigned long long int *parents, int ulonglongRequired, int *mutateIndex, int mutateVal, int chromosomeNo, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    if (max > id)
    {
        for(int i =0;i<mutateVal;i++){
            int index =(blockIdx.x * blockDim.x + threadIdx.x+i)%chromosomeNo;
            int mutateIndexId = mutateIndex[index] / 64 +id;
            int mutateDigit =  mutateIndex[index] % 64; 
            int tmpVar = parents[mutateIndexId];
            if (!((tmpVar >> mutateDigit) & 1))
            {
                tmpVar |= (1ULL << mutateDigit);
            }
            else
            {
                // if chromsome idth index is 1
                tmpVar &= ~(1ULL << mutateDigit);
            }
            parents[mutateIndexId]=  tmpVar;
        }

    }
}
