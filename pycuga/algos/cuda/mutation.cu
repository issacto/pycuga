
#include <hip/hip_runtime.h>
__global__ void mutation(unsigned long long int *parents, int ulonglongRequired, int *mutateIndex, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    if (max > id)
    {

        int mutateIndexId = mutateIndex[blockIdx.x * blockDim.x + threadIdx.x] / 64 +id;
        int mutateDigit =  mutateIndex[blockIdx.x * blockDim.x + threadIdx.x] % 64;
        // if(mutateIndexId<max  && mutateDigit<64 && mutateDigit>=0){
            int tmpVar = parents[mutateIndexId];
            if (!((tmpVar >> mutateDigit) & 1))
            {
                tmpVar |= (1ULL << mutateDigit);
            }
            else
            {
                // if chromsome idth index is 1
                tmpVar &= ~(1ULL << mutateDigit);
            }
            parents[mutateIndexId]=  tmpVar;
        // }

    }
}
