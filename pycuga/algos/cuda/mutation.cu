
#include <hip/hip_runtime.h>
// one-bit and two-bit flip mutation
__global__ void mutation(unsigned long long int *parents, int ulonglongRequired, float *mutateProb, int *mutateIndex, bool mode, bool isMutationKeep, int mutationThreshold, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    int bid = id*ulonglongRequired;
    int mutateIndexId = mutateIndex[(blockIdx.x * blockDim.x + threadIdx.x)]/64+bid;
    int mutateDigit =mutateIndex[(blockIdx.x * blockDim.x + threadIdx.x)]%64;
    if (max > id)
    {
         if (mutateProb[(blockIdx.x * blockDim.x + threadIdx.x)] > mutationThreshold)
        {
            if (!((parents[mutateIndexId] >> mutateDigit) & 1))
            {
                // if chromsome idth index is 0
                parents[mutateIndexId] |= (1ULL << mutateDigit);
            }
            else
            {
                // if chromsome idth index is 1
                parents[mutateIndexId] &= ~(1ULL << mutateDigit);
            }
            parents[mutateIndexId]=11111;
        }
    }
}
