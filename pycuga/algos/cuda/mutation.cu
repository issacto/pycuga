#include "hip/hip_runtime.h"
// one-bit and two-bit flip mutation
__global__ void mutation(unsigned long long int *parents, int ulonglongRequired, float *mutateProb, int *mutateIndex, bool mode, bool isMutationKeep, int mutationThreshold, int max)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x)*ulonglongRequired;
    int bestIndexId = bestIndex*ulonglongRequired;
    int mutateIndexId = mutateIndex[(blockIdx.x * blockDim.x + threadIdx.x)]/64+id;
    int mutateDigit =mutateIndex[(blockIdx.x * blockDim.x + threadIdx.x)]%64;
    if (max > id)
    {
         if (mutateProb[(blockIdx.x * blockDim.x + threadIdx.x)] > mutationThreshold)
        {
            if (!((parents[mutateIndexId] >> mutateDigit) & 1))
            {
                // if chromsome idth index is 0
                parents[mutateIndexId] |= (1ULL << mutateDigit);
            }
            else
            {
                // if chromsome idth index is 1
                parents[mutateIndexId] &= ~(1ULL << mutateDigit);
            }
        }
    }
}
